#include "hip/hip_runtime.h"
#include"matrix.h"
using namespace std;
using namespace std::chrono;
#include <iomanip>
#define LAG 15
#define PATH "/home/jiahuaz/ChangeDetection/test6/"
int main() {

    auto t1 = high_resolution_clock::now();

    //-------------------Read Data------------------------

    double* testMatrix = nullptr;
    double* refMatrices[LAG] ;
    double* as[LAG];
    double* finalRes = nullptr;
    double* rho=nullptr;
    double* maxRho ;
    
    hipMallocManaged(&maxRho,sizeof(double));
    *maxRho=0.0;
    hipMallocManaged(&finalRes,sizeof(double*));
    finalRes=nullptr;

    for(int i=0;i<LAG;++i){
    	hipMallocManaged(&refMatrices[i], SIZE * sizeof(double));
        string path = string(PATH) + "Iref6" + string(1, 'A' + i) + ".dat";
        ReadData(path,refMatrices[i]);
    	hipMallocManaged(&as[i], SIZE * sizeof(double));
	memcpy(as,refMatrix,sizeof(double)*SIZE);	
	}

    hipMallocManaged(&testMatrix, SIZE * sizeof(double));
    ReadData(string(PATH) + "Itest6.dat",testMatrix);
    
    hipMallocManaged(&rho,sizeof(double)*LAG);
    hipMallocManaged(&finalRes, SIZE * sizeof(double));
	
   __global__ Lag(testMatrix,refMatrices,as,finalRes,rho,maxRho){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	double* refMatrix=refMatrices[idx];
	double* a=as[idx];

	Pearson<<<1,512>>>(testMatrix, refMatrix,&rho[idx]);
	hipDeviceSynchronize();
	printf("pearson:%f\n",rho[idx]);

        Add<<<128,512>>>(refMatrix, testMatrix,rho[idx]);
	hipDeviceSynchronize();

        SpatialFilter<<<128,512>>>(refMatrix,a);
	hipDeviceSynchronize();
 
 
        AnomalyDetection(refMatrix);
 
	Pearson<<<1,512>>>(refMatrix, testMatrix,&rho[idx]);
	hipDeviceSynchronize();
   
	if(rho[idx]>*maxRho)
	{
		atomicExch(maxRho,rho[idx);
		atomicExch(finalRes,refMatrix);
	}
   
   }
 
	Lag<<<1,LAG>>>(testMatrix,refMatrices,as,finalRes,rho,maxRho);	

    auto t3 = high_resolution_clock::now();
    duration<double> duration = t3 - t1;
    std::cout << "Compute time: " << duration.count() << " seconds" << std::endl;

    //-----------------------Write Data--------------------
    return 0;
}

