#include"matrix.h"
using namespace std;
using namespace std::chrono;
#include <iomanip>
#define LAG 15
#define PATH "/home/jiahuaz/ChangeDetection/test6/"
int main() {

    auto t1 = high_resolution_clock::now();

    //-------------------Read Data------------------------

    double* testMatrix = nullptr;
    double* refMatrix = nullptr;
    double* finalRes = nullptr;
    double* rho=nullptr;
    double minRho = 1.0;
    double* a=nullptr;
    hipMallocManaged(&rho,sizeof(double));
    hipMallocManaged(&testMatrix, SIZE * sizeof(double));
    hipMallocManaged(&refMatrix, SIZE * sizeof(double));
    hipMallocManaged(&finalRes, SIZE * sizeof(double));
    hipMallocManaged(&a,SIZE*sizeof(double));
    ReadData(string(PATH) + "Itest6.dat",testMatrix);
	
   
    hipStream_t stream[LAG];
    for(int i=0;i<LAG;++i)
	    hipStreamCreate(&stream[i]);
 
    for (int i = 0; i < LAG; ++i) {
        string path = string(PATH) + "Iref6" + string(1, 'A' + i) + ".dat";
        ReadData(path,refMatrix);


        //-----------------------AR(1)-------------------------
	Pearson<<<1,512>>>(testMatrix, refMatrix,rho);
	hipDeviceSynchronize();
	std::cout << "Pearson correlation: " << fixed << std::setprecision(10) << *rho << std::endl;
        Add<<<128,512>>>(refMatrix, testMatrix,*rho);
	hipDeviceSynchronize();
        //-----------SpatialFilter and AnomalyDetection--------
	memcpy(a,refMatrix,sizeof(double)*SIZE);	
        SpatialFilter<<<128,512>>>(refMatrix,a);
	hipDeviceSynchronize();
        AnomalyDetection(refMatrix);

        //-----------update the result--------
	Pearson<<<1,512>>>(refMatrix, testMatrix,rho);
	hipDeviceSynchronize();
        if (*rho < minRho) {
            minRho = *rho;
            memcpy(finalRes, refMatrix, sizeof(double) * SIZE);
        }


    }

    auto t3 = high_resolution_clock::now();
    duration<double> duration = t3 - t1;
    std::cout << "Compute time: " << duration.count() << " seconds" << std::endl;

    for(int i=0;i<LAG;++i)
	    hipStreamDestroy(&stream[i]);

    //-----------------------Write Data--------------------
    return 0;
}

