#include "hip/hip_runtime.h"
#include"matrix.h"
using namespace std;
using namespace std::chrono;
#include <iomanip>
#define LAG 18
#define PATH "/home/jiahuaz/ChangeDetection/test6/"
__global__ void LagPearson(double* testMatrix,double**refMatrices,double* rho){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	double* refMatrix=refMatrices[idx];
	Pearson<<<1,512>>>(testMatrix, refMatrix,&rho[idx]);
	hipError_t err = hipGetLastError();
   }
 
__global__ void LagAdd(double* testMatrix,double**refMatrices,double* rho){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	double* refMatrix=refMatrices[idx];
	Add<<<128,512>>>(refMatrix,testMatrix,rho[idx]);
   }
__global__ void LagSF(double**refMatrices,double** as){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	double* refMatrix=refMatrices[idx];
	double* a=as[idx];
	SpatialFilter<<<128,512>>>(refMatrix,a);
   }
int main() {


	//-------------------Read Data------------------------
	//testMatrix
	double* testMatrix = nullptr;
	hipMallocManaged(&testMatrix, SIZE * sizeof(double));
	ReadData(string(PATH) + "Itest6.dat",testMatrix);
	
	//refMatrices and their copies
	double** refMatrices = nullptr;
	double** as=nullptr;//A copy of refMatrices. Spatial filtering directly modififies refMatrices, and as[LAG] serves as the original .
	hipMallocManaged(&refMatrices,SIZE*sizeof(double*));
	hipMallocManaged(&as,SIZE*sizeof(double*));
	for(int i=0;i<LAG;++i){
		hipMallocManaged(&refMatrices[i], SIZE * sizeof(double));
		string path = string(PATH) + "Iref6" + string(1, 'A' + i) + ".dat";
		ReadData(path,refMatrices[i]);
		hipMallocManaged(&as[i], SIZE * sizeof(double));
		memcpy(as[i],refMatrices[i],sizeof(double)*SIZE);	
	}
	
	//rho	
	double* rho=nullptr;
	hipMallocManaged(&rho,sizeof(double)*LAG);
	
	auto t1 = high_resolution_clock::now();
        //--------------------Launch Kernel--------------------
	LagPearson<<<1,LAG>>>(testMatrix,refMatrices,rho);	
	hipDeviceSynchronize();
	LagAdd<<<1,LAG>>>(testMatrix,refMatrices,rho);	
	hipDeviceSynchronize();
	LagSF<<<1,LAG>>>(refMatrices,as);	
	hipDeviceSynchronize();
	for(int i=0;i<LAG;++i)
		AnomalyDetection(refMatrices[i]);
	LagPearson<<<1,LAG>>>(testMatrix,refMatrices,rho);	
	hipDeviceSynchronize();
	
	
	auto t3 = high_resolution_clock::now();
        //-----------------------Write Data--------------------
/*	double maxRho=0;
	double maxIdx=-1;
	for(int i=0;i<LAG;++i)
	{
		if(rho[i]>maxRho){
			maxRho=rho[i];
			maxIdx=i;
		}
	}
	writeData("res.txt",refMatrices[maxIdx]);
*/		

	
	//---------------------Memory Free---------------------
	

	//---------------------Compute Time--------------------
	duration<double> duration = t3 - t1;
	std::cout << "Compute time: " << duration.count() << " seconds" << std::endl;
	return 0;
}

