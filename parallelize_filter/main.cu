#include"matrix.h"
using namespace std;
using namespace std::chrono;
#include <iomanip>
#define LAG 1
#define PATH "/home/jiahuaz/ChangeDetection/test6/"
int main() {

    auto t1 = high_resolution_clock::now();

    //-------------------Read Data------------------------

    double* testMatrix = nullptr;
    double* refMatrix = nullptr;
    double* finalRes = nullptr;
    double* rho=nullptr;
    double minRho = 1.0;
    double* a=nullptr;
    hipMallocManaged(&rho,sizeof(double));
    hipMallocManaged(&testMatrix, SIZE * sizeof(double));
    hipMallocManaged(&refMatrix, SIZE * sizeof(double));
    hipMallocManaged(&finalRes, SIZE * sizeof(double));
    hipMallocManaged(&a,SIZE*sizeof(double));
    ReadData(string(PATH) + "Itest6.dat",testMatrix);

/*     for(int i=0;i<SIZE;++i)
	     refMatrix[i]=1+i;

     memcpy(a,refMatrix,SIZE*sizeof(double));
     SpatialFilter<<<64,512>>>(refMatrix,a);
     hipDeviceSynchronize();

	writeData("./res.txt",refMatrix);
*/

 
    for (int i = 0; i < LAG; ++i) {
        string path = string(PATH) + "Iref6" + string(1, 'A' + i) + ".dat";
        ReadData(path,refMatrix);


        //-----------------------AR(1)-------------------------
	Pearson<<<1,512>>>(testMatrix, refMatrix,rho);
	hipDeviceSynchronize();
	std::cout << "Pearson correlation: " << fixed << std::setprecision(10) << *rho << std::endl;
        Add(refMatrix, testMatrix,*rho);
        //-----------SpatialFilter and AnomalyDetection--------
	memcpy(a,refMatrix,sizeof(double)*SIZE);	
        SpatialFilter<<<128,512>>>(refMatrix,a);
	hipDeviceSynchronize();
	writeData("./res.txt",refMatrix);
        AnomalyDetection(refMatrix);

        //-----------update the result--------
	Pearson<<<1,512>>>(refMatrix, testMatrix,rho);
	hipDeviceSynchronize();
        if (*rho < minRho) {
            minRho = *rho;
            memcpy(finalRes, refMatrix, sizeof(double) * SIZE);
        }


    }

    auto t3 = high_resolution_clock::now();
    duration<double> duration = t3 - t1;
    std::cout << "Compute time: " << duration.count() << " seconds" << std::endl;

    //-----------------------Write Data--------------------
    return 0;
}

